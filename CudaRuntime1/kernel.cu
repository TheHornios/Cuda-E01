﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Entrega 1 
*
* Alumno: Rodrigo Pascual Arnaiz y Villar Solla, Alejandro
* Fecha: 02/11/2022
*
*/

///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

///////////////////////////////////////////////////////////////////////////
// defines
#define M 7
#define N 25
///////////////////////////////////////////////////////////////////////////
// declaracion de funciones
// HOST: funcion llamada desde el host y ejecutada en el host
/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
* es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: cudaDeviceProp -> retorna el onjeto que tiene todas las
* propiedades del dispositivo CUDA
*/
__host__ void propiedadesDispositivo(int id_dispositivo)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, id_dispositivo);
	// calculo del numero de cores (SP)
	int cuda_cores = 0;
	int multi_processor_count = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	char* arquitectura = (char*)"";
	switch (major)
	{
	case 1:
		//TESLA
		cuda_cores = 8;
		arquitectura = (char*)"TESLA";
		break;
	case 2:
		//FERMI
		arquitectura = (char*)"FERMI";
		if (minor == 0)
			cuda_cores = 32;
		else
			cuda_cores = 48;
		break;
	case 3:
		//KEPLER
		arquitectura = (char*)"KEPLER";
		cuda_cores = 192;
		break;
	case 5:
		//MAXWELL
		arquitectura = (char*)"MAXWELL";
		cuda_cores = 128;
		break;
	case 6:
		//PASCAL
		arquitectura = (char*)"PASCAL";
		cuda_cores = 64;
		break;
	case 7:
		//VOLTA
		arquitectura = (char*)"VOLTA";
		cuda_cores = 64;
		break;
	case 8:
		//AMPERE
		arquitectura = (char*)"AMPERE";
		cuda_cores = 128;
		break;
	default:
		arquitectura = (char*)"DESCONOCIDA";
		//DESCONOCIDA
		cuda_cores = 0;
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	int rtV;
	hipRuntimeGetVersion(&rtV);
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", id_dispositivo, deviceProp.name);
	printf("***************************************************\n");
	printf("> CUDA Toolkit \t\t\t\t: %d.%d\n", rtV / 1000, (rtV % 1000) / 10);
	printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
	printf("> Arquitectura CUDA \t\t\t: %s \n", arquitectura);
	printf("> No. de MultiProcesadores \t\t: %d \n",
		multi_processor_count);
	printf("> No. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores,
		multi_processor_count, cuda_cores *
		multi_processor_count);
	printf("> No. max. de Hilos (por bloque) \t: %d \n",
		deviceProp.maxThreadsPerBlock);
	printf("> Memoria Global (total) \t\t: %zu MiB\n",
		deviceProp.totalGlobalMem / (1 << 20));

	printf("***************************************************\n");
	printf("> KERNEL DE %i BLOQUE con %i HILOS:\n", 1, N * M);
	printf("\teje x -> %i hilos\n", N);
	printf("\teje y -> %i hilos\n", M);
	
}

///////////////////////////////////////////////////////////////////////////
// HOST: funcion llamada desde el host y ejecutada en el host
/**
* Funcion: rellenarVectorHst
* Objetivo: Funcion que rellena un array pasado por parametro
* con numero aleatorios del 1 al 9
*
* Param: INT* arr -> Puntero del array a rellenar
* Return: void
*/
__host__ void rellenarVectorHst(int* arr)
{

	

	for (size_t i = 0; i < M; i++)
	{
		int num_aleatorio = rand() % 10;
		for (size_t t = 0; t < N; t++)
		{

			arr[N * i + t] = num_aleatorio;
		}
	}
}

///////////////////////////////////////////////////////////////////////////
// KERNEL: Función que deja las columnas impares a 0
/**
* Funcion: desplazarAbajo
* Objetivo: Funcion que desplaza una una fila completa de un array una posicion mas abajo 
*  y mueve el ultimo elemento a la primera posicion
* 
* Param: INT* arr -> Puntero del array que tiene los datos
* Param: INT* arr_final -> Puntero del array a rellenar
* Return: void
*/

__global__ void desplazarAbajo(int* arr, int* arr_final )
{
	int columna = threadIdx.x;
	int fila = threadIdx.y;
	int pos = fila * N + columna;


	if (fila == 0) {
		arr_final[pos] = arr[( M - 1 ) * N + columna];
	}
	else {
		arr_final[pos] = arr[( fila - 1 )* N + columna];
	}

}
///////////////////////////////////////////////////////////////////////////
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	srand(time(NULL));
	// buscando dispositivos
	int numero_dispositivos;
	hipGetDeviceCount(&numero_dispositivos);
	if (numero_dispositivos != 0)
	{
		for (int i = 0; i < numero_dispositivos; i++)
		{
			propiedadesDispositivo(i);
		}
	}
	else
	{
		printf("!!!!!ERROR!!!!!\n");
		printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}

	// Básico 5
	// Declaración de variables
	int* hst_original, * hst_final;
	int* dev_original, * dev_final;
	// Declaración de eventos
	hipEvent_t start;
	hipEvent_t stop;
	// Asignación de espacio a las variables en el host
	hst_original = (int*)malloc(N * M * sizeof(int));
	hst_final = (int*)malloc(N * M * sizeof(int));

	// Asignación de espacio a las variables en el device
	hipMalloc((void**)&dev_original, N * M * sizeof(int));
	hipMalloc((void**)&dev_final, N * M * sizeof(int));
	// Creación de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Rellenar con información
	

	rellenarVectorHst(hst_original );

	// Copiar datos al dispositivo
	hipMemcpy(dev_original, hst_original, sizeof(int) * N * M,
		hipMemcpyHostToDevice);
	// Desplazar filas
	dim3 blocks(1);
	dim3 threads(N, M);
	//// Marca de inicio
	hipEventRecord(start, 0);
	//// Función KERNEL
	desplazarAbajo <<<blocks, threads >>> (dev_original, dev_final);
	//// Marca de fin
	hipEventRecord(stop, 0);
	//// Sincronizar Eventos
	hipEventSynchronize(stop);
	// Traer datos del device
	hipMemcpy(hst_final, dev_final, sizeof(int) * N * M,
		hipMemcpyDeviceToHost);
	// Mostrar tiempo de ejecución, original y resultado

	//// Calcular tiempo
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("> Tiempo de ejecucion: %0.6f ms\n", elapsedTime);
	printf("> MATRIZ ORIGINAL:\n");
	for (int y = 0; y < M; y++)
	{
		for (int x = 0; x < N; x++)
		{
			printf("%i  ", hst_original[N * y + x]);
		}
		printf("\n");
	}

	printf("\n");
	printf("> MATRIZ FINAL:\n");
	for (int y = 0; y < M; y++) 
	{
		for (int x = 0; x < N; x++)
		{
			printf("%i  ", hst_final[N * y + x]);
		}
		printf("\n");
	}
	// Salida del programa
	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////